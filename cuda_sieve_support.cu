#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdint>
#include <string>
#include <stdexcept>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__global__ void initialize_sieve_interval(uint64_t* storage, uint64_t lb, int length) {
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if (id < length) {
		storage[id] = id + lb;
	}
};

#define COMPOSITE 0
#define MAX_STORAGE_SIZE 1024 * 1024 * 100
enum SearchIntervalState { SIS_NEW, SIS_READY, SIS_ACTIVE, SIS_INCONSISTENT };
class CudaSearchInterval {
		uint64_t  lb;
		uint64_t  ub;
		uint64_t  population;
		uint64_t  capacity;
		uint64_t* internal_storage;
		SearchIntervalState state;
		void set_extrema_safely(uint64_t lowerbound, uint64_t upperbound) {
			uint64_t potential_population = upperbound - lowerbound;
			if (potential_population > MAX_STORAGE_SIZE) {
				upperbound = lowerbound + MAX_STORAGE_SIZE - 1;
			}
			
			lb = lowerbound;
			ub = upperbound;
			capacity = MAX_STORAGE_SIZE;
			population = ub - lb;
			state = SIS_INCONSISTENT;
		};
	public:
		CudaSearchInterval(uint64_t lowerbound, uint64_t upperbound) 
		{
			state = SIS_NEW;
			set_extrema_safely(lowerbound, upperbound);
			hipError_t err = hipMalloc(&internal_storage, capacity * sizeof(uint64_t));
			if(err != hipSuccess) {
				std::cout << "Couldn't actuate space memory on the Jeep U. Everything is fucked, Bubbs!" << std::endl;
			} else {
				initialize();
			}
		};
		~CudaSearchInterval() {
			hipFree(internal_storage);
		};
		void repopulate(uint64_t lowerbound, uint64_t upperbound) {
			set_extrema_safely(lowerbound, upperbound);
			initialize();
		};
		void initialize() {
			initialize_sieve_interval(internal_storage, lb, population);
			state = SIS_READY;
		};
		uint64_t lowerbound() const {
			return lb;
		};
		uint64_t upperbound() const {
			return ub;
		};
		uint64_t smallest_multiple(uint64_t prime) {
			uint64_t residue = (lb % prime);
			if (residue == 0) {
				// lowerbound is a multiple of prime
				return lb;
			} else {
				// lowerbound - residue is a multiple of prime
				// but it lies outside the range, so increment by prime
				return lb - residue + prime;
			}	
		};
		uint64_t largest_multiple(uint64_t prime) {
			// If residue is 7, then upperbound is 7 greater
			// than largest multiple of prime in interval
			uint64_t residue = ub % prime;
			return ub - residue;
		}
		void mark_composite(uint64_t composite) {
			uint64_t index = composite - lb;
			if (index <= population) {
				internal_storage[index] = COMPOSITE;
			} else {
				std::cout << "Attempted to access composite outside of bounds" << std::endl;
			}
		};
		std::vector<uint64_t> get_primes() {
			std::vector<uint64_t> primes;
			for(uint64_t i = 0; i < population; i++) {
				uint64_t candidate = internal_storage[i];
				if(candidate != COMPOSITE) {
					primes.push_back(candidate);
				}
			}
			return primes;
		};
		void mark_multiples_of_prime(uint64_t prime) {
			// Determine smallest composite in range (start)
			uint64_t start = smallest_multiple(prime);

			// Determine largest composite in range (stop)
			uint64_t stop = largest_multiple(prime);

			// Mark all multiples of prime
			for(uint64_t composite = start; composite <= stop; composite += prime) {
				mark_composite(composite);
			}
		};
		void apply_sieve(std::vector<uint64_t> primes) {
			state = SIS_ACTIVE;
			for(int i = 0; i < primes.size(); i++) {
				uint64_t p = primes.at(i);
				mark_multiples_of_prime(p);
			}
		};
		friend std::ostream& operator<< (std::ostream& o, const CudaSearchInterval& si) {
				return o << "[" << si.lowerbound() << "," << si.upperbound() << "]";
		};
};


